// 
// Mex wrapper to CUSPARSE format converter (csr2coo).
//
// Inspired by cusparse samples (conugateGradient) and Matlab gcsparse.
//  http://docs.nvidia.com/cuda/cusparse/index.html#cusparse-lt-t-gt-csrmv
//  http://www.mathworks.com/matlabcentral/fileexchange/44423-gpu-sparse--accumarray--non-uniform-grid
//

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

/* Using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

// MATLAB related
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "mxShowCriticalErrorMessage.c"

// Input Arguments
#define	ROW_CSR prhs[0]
#define	NROWS   prhs[1]

// Output Arguments
#define	ROW plhs[0]

void mexFunction(int nlhs, mxArray * plhs[], int nrhs, const mxArray * prhs[])
{
    // Checks
    if (nlhs > 1) mxShowCriticalErrorMessage("wrong number of output arguments",nlhs);
    if (nrhs != 2) mxShowCriticalErrorMessage("wrong number of input arguments",nrhs);

    // Initialize the MathWorks GPU API
    mxInitGPU();

    // Create Matlab pointers on the GPU
    mxGPUArray const *row_csr = mxGPUCreateFromMxArray(ROW_CSR);

    // Checks - note rows must be in CSR format
    if (!mxIsScalar(NROWS)) mxShowCriticalErrorMessage("NROWS argument must be a scalar");
    if (mxGPUGetClassID(row_csr) != mxINT32_CLASS) mxShowCriticalErrorMessage("ROW_CSR argument is not int32");
    mwSize nrows = mxGetScalar(NROWS);
    if (mxGPUGetNumberOfElements(row_csr) != nrows+1) mxShowCriticalErrorMessage("ROW_CSR argument is wrong size",mxGPUGetNumberOfElements(row_csr));

    // Get handle to the CUBLAS context
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) mxShowCriticalErrorMessage(hipblasStatus_t);

    // Get handle to the CUSPARSE context
    hipsparseStatus_t status;
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&cusparseHandle);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) mxShowCriticalErrorMessage(cusparseStatus);
    hipsparseMatDescr_t descr = 0;
    cusparseStatus = hipsparseCreateMatDescr(&descr);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) mxShowCriticalErrorMessage(cusparseStatus);
    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ONE);

    // Convert from matlab pointers to native pointers 
    const int * const d_row_csr = (int*)mxGPUGetDataReadOnly(row_csr);

    // Now we can access the arrays, we can do some checks
    int base;
    hipMemcpy(&base, d_row_csr, sizeof(int), hipMemcpyDeviceToHost);
    if (base != HIPSPARSE_INDEX_BASE_ONE) mxShowCriticalErrorMessage("ROW_CSR not using 1-based indexing");

    int nnz;
    hipMemcpy(&nnz, d_row_csr+nrows, sizeof(int), hipMemcpyDeviceToHost);
    nnz -= HIPSPARSE_INDEX_BASE_ONE;
    if (nnz < 0) mxShowCriticalErrorMessage("ROW_CSR returned negative nnz");

    // Create space for output vector
    const mwSize ndim = 1;
    mwSize dims[ndim] = {(mwSize)nnz}; // we checked that nnz is >=0 so cast is safe
    mxGPUArray *row = mxGPUCreateGPUArray(ndim, dims, mxINT32_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    if (row==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");

    // Convert from matlab pointers to native pointers 
    int *d_row = (int*)mxGPUGetData(row);

    // Call csr2coo
    status = hipsparseXcsr2coo(cusparseHandle, d_row_csr, nnz, nrows, d_row, HIPSPARSE_INDEX_BASE_ONE);

    if (status == HIPSPARSE_STATUS_SUCCESS)
    {
    	// Return result
    	ROW = mxGPUCreateMxArrayOnGPU(row);

    	// Make sure operations are finished before deleting
    	//hipDeviceSynchronize();
    }

    // Clean up
    hipsparseDestroyMatDescr(descr);
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);
    mxGPUDestroyGPUArray(row);
    mxGPUDestroyGPUArray(row_csr);

    // Failure
    if (status != HIPSPARSE_STATUS_SUCCESS)
    {
	mxShowCriticalErrorMessage("Operation hipsparseXcsr2coo failed",status);
    }

    return;
}
