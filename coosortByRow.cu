// 
// Mex wrapper to CUSPARSE sort for COO format (coosortByRow).
//
// Inspired by cusparse samples (conugateGradient) and Matlab gcsparse.
//  http://docs.nvidia.com/cuda/cusparse/index.html#cusparse-lt-t-gt-csrmv
//  http://www.mathworks.com/matlabcentral/fileexchange/44423-gpu-sparse--accumarray--non-uniform-grid
//

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

/* Using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

// MATLAB related
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "mxShowCriticalErrorMessage.c"

// Input Arguments
#define	ROW   prhs[0]
#define	COL   prhs[1]
#define	VAL   prhs[2]
#define	NROWS prhs[3]
#define	NCOLS prhs[4]

// Output Arguments
#define	ROW_SORT plhs[0]
#define	COL_SORT plhs[1]
#define	VAL_SORT plhs[2]

void mexFunction(int nlhs, mxArray * plhs[], int nrhs, const mxArray * prhs[])
{
    // Checks
    if (nlhs > 3) mxShowCriticalErrorMessage("wrong number of output arguments",nlhs);
    if (nrhs != 5) mxShowCriticalErrorMessage("wrong number of input arguments",nrhs);

    // Initialize the MathWorks GPU API
    mxInitGPU();

    // Create Matlab pointers on the GPU
    mxGPUArray const *row = mxGPUCreateFromMxArray(ROW);
    mxGPUArray const *col = mxGPUCreateFromMxArray(COL);
    mxGPUArray const *val = mxGPUCreateFromMxArray(VAL);

    // Checks - note vectors must be in COO (uncompressed) format
    int nnz = mxGPUGetNumberOfElements(val);
    if (mxGPUGetNumberOfElements(row) != nnz) mxShowCriticalErrorMessage("ROW and VAL argument length mismatch");
    if (mxGPUGetNumberOfElements(col) != nnz) mxShowCriticalErrorMessage("COL and VAL argument length mismatch");

    if (!mxIsScalar(NROWS)) mxShowCriticalErrorMessage("NROWS argument must be a scalar");
    if (!mxIsScalar(NCOLS)) mxShowCriticalErrorMessage("NCOLS argument must be a scalar");

    if (mxGPUGetClassID(row) != mxINT32_CLASS) mxShowCriticalErrorMessage("ROW argument is not int32");
    if (mxGPUGetClassID(col) != mxINT32_CLASS) mxShowCriticalErrorMessage("COL argument is not int32");
    if (mxGPUGetClassID(val) != mxSINGLE_CLASS) mxShowCriticalErrorMessage("VAL argument is not single");

    int nrows = (int)mxGetScalar(NROWS);
    int ncols = (int)mxGetScalar(NCOLS);

    // Create space for output vectors
    const mwSize ndim = 1;
    mwSize dims[ndim];

    dims[0] = nnz;
    mxGPUArray *row_sort = mxGPUCreateGPUArray(ndim, dims, mxINT32_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    if (row_sort==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");

    mxGPUArray *col_sort = mxGPUCreateGPUArray(ndim, dims, mxINT32_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    if (col_sort==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");

    mxComplexity ccx = mxGPUGetComplexity(val);
    mxGPUArray *val_sort = mxGPUCreateGPUArray(ndim, dims, mxSINGLE_CLASS, ccx, MX_GPU_INITIALIZE_VALUES);
    if (val_sort==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");

    // Get handle to the CUBLAS context
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) mxShowCriticalErrorMessage(hipblasStatus_t);

    // Get handle to the CUSPARSE context
    hipError_t cudaStatus;
    hipsparseStatus_t cusparseStatus;
    hipsparseHandle_t cusparseHandle = 0;
    cusparseStatus = hipsparseCreate(&cusparseHandle);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) mxShowCriticalErrorMessage(cusparseStatus);
    hipsparseMatDescr_t descr = 0;
    cusparseStatus = hipsparseCreateMatDescr(&descr);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) mxShowCriticalErrorMessage(cusparseStatus);
    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ONE);

    // Convert from matlab pointers to native pointers 
    const int * const d_row = (int*)mxGPUGetDataReadOnly(row);
    const int * const d_col = (int*)mxGPUGetDataReadOnly(col);
    int *d_col_sort = (int*)mxGPUGetData(col_sort);
    int *d_row_sort = (int*)mxGPUGetData(row_sort);

    // Since sort is in-place, copy the read-only vectors to the read-write ones
    cudaStatus = hipMemcpy((void *)d_row_sort, d_row, nnz*sizeof(int), hipMemcpyDeviceToDevice);
    if (cudaStatus != hipSuccess) mxShowCriticalErrorMessage("Operation hipMemcpy failed",cudaStatus);

    cudaStatus = hipMemcpy((void *)d_col_sort, d_col, nnz*sizeof(int), hipMemcpyDeviceToDevice);
    if (cudaStatus != hipSuccess) mxShowCriticalErrorMessage("Operation hipMemcpy failed",cudaStatus);

    if (ccx == mxREAL)
    {
    	const float * const d_val = (float*)mxGPUGetDataReadOnly(val);
    	float *d_val_sort = (float*)mxGPUGetData(val_sort);
    	cudaStatus = hipMemcpy((void *)d_val_sort, d_val, nnz*sizeof(float), hipMemcpyDeviceToDevice);
    }
    else
    {
    	const hipFloatComplex * const d_val = (hipFloatComplex*)mxGPUGetDataReadOnly(val);
    	hipFloatComplex *d_val_sort = (hipFloatComplex*)mxGPUGetData(val_sort);
    	cudaStatus = hipMemcpy((void *)d_val_sort, d_val, nnz*sizeof(hipFloatComplex), hipMemcpyDeviceToDevice);
    }
    if (cudaStatus != hipSuccess) mxShowCriticalErrorMessage("Operation hipMemcpy failed",cudaStatus);

    // Sort by rows
    int *P = NULL;
    void *pBuffer = NULL;
    size_t pBufferSizeInBytes = 0;

    if (nnz > 0)
    {
    	// step 1: allocate buffer
    	cusparseStatus = hipsparseXcoosort_bufferSizeExt(cusparseHandle, nrows, ncols, nnz, d_row, d_col, &pBufferSizeInBytes);
    	if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) mxShowCriticalErrorMessage("Operation hipsparseXcoosort_bufferSizeExt failed",cusparseStatus);

    	cudaStatus = hipMalloc( &pBuffer, sizeof(char)*pBufferSizeInBytes);
    	if (cudaStatus != hipSuccess) mxShowCriticalErrorMessage("Operation hipMalloc failed",cudaStatus);

    	// step 2: setup permutation vector P to identity
    	cudaStatus = hipMalloc( &P, sizeof(int)*nnz);
    	if (cudaStatus != hipSuccess) mxShowCriticalErrorMessage("Operation hipMalloc failed",cudaStatus);

        cusparseStatus = hipsparseCreateIdentityPermutation(cusparseHandle, nnz, P);
    	if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) mxShowCriticalErrorMessage("Operation hipsparseCreateIdentityPermutation failed",cusparseStatus);

    	// step 3: sort COO format by Row
    	cusparseStatus = hipsparseXcoosortByRow(cusparseHandle, nrows, ncols, nnz, d_row_sort, d_col_sort, P, pBuffer);
    	if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) mxShowCriticalErrorMessage("Operation hipsparseXcoosortByRow failed",cusparseStatus);

    	// step 4: gather sorted cooVals
    	if (ccx == mxREAL)
    	{
    	    const float * const d_val = (float*)mxGPUGetDataReadOnly(val);
    	    float *d_val_sort = (float*)mxGPUGetData(val_sort);
            cusparseStatus = hipsparseSgthr(cusparseHandle, nnz, d_val, d_val_sort, P, HIPSPARSE_INDEX_BASE_ZERO); // MUST USE BASE_ZERO
        }
        else
        {
    	    const hipFloatComplex * const d_val = (hipFloatComplex*)mxGPUGetDataReadOnly(val);
    	    hipFloatComplex *d_val_sort = (hipFloatComplex*)mxGPUGetData(val_sort);
            cusparseStatus = hipsparseCgthr(cusparseHandle, nnz, d_val, d_val_sort, P, HIPSPARSE_INDEX_BASE_ZERO); // MUST USE BASE_ZERO
        }
    	if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) mxShowCriticalErrorMessage("Operation hipsparseSgthr or hipsparseCgthr failed",cusparseStatus);

    }

    // Return result
    ROW_SORT = mxGPUCreateMxArrayOnGPU(row_sort);
    COL_SORT = mxGPUCreateMxArrayOnGPU(col_sort);
    VAL_SORT = mxGPUCreateMxArrayOnGPU(val_sort);

    // Make sure operations are finished before deleting
    //hipDeviceSynchronize();

    // Clean up
    hipsparseDestroyMatDescr(descr);
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);
    mxGPUDestroyGPUArray(row);
    mxGPUDestroyGPUArray(row_sort);
    mxGPUDestroyGPUArray(col);
    mxGPUDestroyGPUArray(col_sort);
    mxGPUDestroyGPUArray(val);
    mxGPUDestroyGPUArray(val_sort);
    if (pBuffer) hipFree(pBuffer);
    if (P) hipFree(P);

    return;
}
