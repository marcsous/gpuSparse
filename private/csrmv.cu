// 
// Mex wrapper to CUSPARSE matrix-vector multiply (csrmv).
//
// Inspired by cusparse samples (conugateGradient) and Matlab gcsparse.
//  http://docs.nvidia.com/cuda/cusparse/index.html#cusparse-lt-t-gt-csrmv
//  http://www.mathworks.com/matlabcentral/fileexchange/44423-gpu-sparse--accumarray--non-uniform-grid
//

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <iostream>

/* Using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

#if CUDART_VERSION >= 11000
#include "wrappers_to_cuda_11.h"
#endif
        
// MATLAB related
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "mxShowCriticalErrorMessage.c"

// Input Arguments
#define	ROW_CSR prhs[0] // this in CSR format (returned from coo2csr.cu)
#define	COL     prhs[1]
#define	VAL     prhs[2]
#define	NROWS   prhs[3]
#define	NCOLS   prhs[4]
#define	TRANS   prhs[5]
#define	X       prhs[6]

// Output Arguments
#define	Y	plhs[0]

void mexFunction(int nlhs, mxArray * plhs[], int nrhs, const mxArray * prhs[])
{
    // Checks
    if (nlhs > 1) mxShowCriticalErrorMessage("wrong number of output arguments",nlhs);
    if (nrhs != 7) mxShowCriticalErrorMessage("wrong number of input arguments",nrhs);

    if(!mxIsGPUArray(ROW_CSR)) mxShowCriticalErrorMessage("ROW_CSR argument is not on GPU");
    if(!mxIsGPUArray(COL)) mxShowCriticalErrorMessage("COL argument is not on GPU");
    if(!mxIsGPUArray(VAL)) mxShowCriticalErrorMessage("VAL argument is not on GPU");
    if(!mxIsGPUArray(X)) mxShowCriticalErrorMessage("X argument is not on GPU");

    if (!mxIsScalar(NROWS)) mxShowCriticalErrorMessage("NROWS argument must be a scalar");
    if (!mxIsScalar(NCOLS)) mxShowCriticalErrorMessage("NCOLS argument must be a scalar");
    if (!mxIsScalar(TRANS)) mxShowCriticalErrorMessage("TRANS argument must be a scalar");

    // Initialize the MathWorks GPU API
    mxInitGPU();

    // Create Matlab pointers on the GPU
    mxGPUArray const *row_csr = mxGPUCreateFromMxArray(ROW_CSR);
    mxGPUArray const *col = mxGPUCreateFromMxArray(COL);
    mxGPUArray const *val = mxGPUCreateFromMxArray(VAL);
    mxGPUArray const *x = mxGPUCreateFromMxArray(X);

    // Check sizes - note rows are in CSR (compressed row) format
    mwSize nnz = mxGPUGetNumberOfElements(val);
    mwSize nrows = mxGetScalar(NROWS);
    mwSize ncols = mxGetScalar(NCOLS);

    mwSize *xdims = (mwSize*)mxGPUGetDimensions(x); // xdims always has >= 2 elements
    if (mxGPUGetNumberOfDimensions(x) > 2) mxShowCriticalErrorMessage("X argument has too many dimensions",mxGPUGetNumberOfDimensions(x));
    if (xdims[1] != 1) mxShowCriticalErrorMessage("X argument is not a column vector");

    int nx = xdims[0];

    if (mxGPUGetNumberOfElements(row_csr) != nrows+1) mxShowCriticalErrorMessage("ROW_CSR argument wrong size",mxGPUGetNumberOfElements(row_csr));
    if (mxGPUGetNumberOfElements(col) != nnz) mxShowCriticalErrorMessage("COL argument wrong size",mxGPUGetNumberOfElements(col));

    hipsparseOperation_t trans = (hipsparseOperation_t)mxGetScalar(TRANS);
    if (trans == HIPSPARSE_OPERATION_NON_TRANSPOSE)
    {
	if (nx != ncols) mxShowCriticalErrorMessage("X argument wrong size for multiply",nx);
    }
    else
    {
	if (nx != nrows) mxShowCriticalErrorMessage("X argument wrong size for transpose multiply",nx);
    }

    // Check types
    if (mxGPUGetClassID(row_csr) != mxINT32_CLASS) mxShowCriticalErrorMessage("ROW_CSR argument is not int32");
    if (mxGPUGetClassID(col) != mxINT32_CLASS) mxShowCriticalErrorMessage("COL argument is not int32");
    if (mxGPUGetClassID(val) != mxSINGLE_CLASS) mxShowCriticalErrorMessage("VAL argument is not single");
    if (mxGPUGetClassID(x) != mxSINGLE_CLASS) mxShowCriticalErrorMessage("X argument is not single");

    // Check real/complex - mixed is not supported except special case (real A / complex x)
    mxComplexity ccx = mxGPUGetComplexity(x);
    mxComplexity ccv = mxGPUGetComplexity(val);
    mxComplexity ccy = (ccx==mxCOMPLEX || ccv==mxCOMPLEX) ? mxCOMPLEX : mxREAL;
    if(ccx==mxREAL && ccv==mxCOMPLEX) mxShowCriticalErrorMessage("Complex matrix and real vector not supported");
            
    // Create space for output vector
    const mwSize ndim = 1;
    mwSize dims[ndim] = {trans == HIPSPARSE_OPERATION_NON_TRANSPOSE ? nrows : ncols};
    mxClassID cid = mxGPUGetClassID(x);
    mxGPUArray *y;

    // Get handle to the CUBLAS context
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) mxShowCriticalErrorMessage(hipblasStatus_t);

    // Get handle to the CUSPARSE context
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&cusparseHandle);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) mxShowCriticalErrorMessage(cusparseStatus);
    hipsparseMatDescr_t descr = 0;
    cusparseStatus = hipsparseCreateMatDescr(&descr);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) mxShowCriticalErrorMessage(cusparseStatus);
    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ONE);

    // Convert from matlab pointers to native pointers
    const int* const d_row_csr = (int*)mxGPUGetDataReadOnly(row_csr);
    const int* const d_col = (int*)mxGPUGetDataReadOnly(col);

    // Now we can access the arrays, we can do some checks
    int base;
    hipMemcpy(&base, d_row_csr, sizeof(int), hipMemcpyDeviceToHost);
    if (base != HIPSPARSE_INDEX_BASE_ONE) mxShowCriticalErrorMessage("ROW_CSR not using 1-based indexing");

    int nnz_check;
    hipMemcpy(&nnz_check, d_row_csr+nrows, sizeof(int), hipMemcpyDeviceToHost);
    nnz_check -= HIPSPARSE_INDEX_BASE_ONE;
    if (nnz_check != nnz) mxShowCriticalErrorMessage("ROW_CSR argument last element != nnz",nnz_check);

    // Call cusparse multiply function in (S)ingle precision
    if (ccv==mxREAL && ccx==mxREAL)
    {
        const float alpha = 1.0; 
        const float beta = 0.0;
        y = mxGPUCreateGPUArray(ndim, dims, cid, ccy, MX_GPU_INITIALIZE_VALUES);
        if (y==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed.");
        float* d_y = (float*)mxGPUGetData(y);
    	const float* const d_val = (float*)mxGPUGetDataReadOnly(val);
    	const float* const d_x = (float*)mxGPUGetDataReadOnly(x);
#if CUDART_VERSION >= 11000
        cusparseStatus = cusparseXcsrmv_wrapper(cusparseHandle, trans, nrows, ncols, nnz, &alpha, descr, d_val, d_row_csr, d_col, d_x, &beta, d_y);
#else
        cusparseStatus = hipsparseScsrmv(cusparseHandle, trans, nrows, ncols, nnz, &alpha, descr, d_val, d_row_csr, d_col, d_x, &beta, d_y);
#endif
    }
    else if (ccv==mxREAL && ccx==mxCOMPLEX)
    {
        const float alpha = 1.0; 
        const float beta = 0.0;
        const float* const d_val = (float*)mxGPUGetDataReadOnly(val);

        mxGPUArray* y_real = mxGPUCreateGPUArray(ndim, dims, cid, mxREAL, MX_GPU_INITIALIZE_VALUES);
        mxGPUArray* y_imag = mxGPUCreateGPUArray(ndim, dims, cid, mxREAL, MX_GPU_INITIALIZE_VALUES);
        if(!y_real || !y_imag) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed.");
        float* d_y_real = (float*)mxGPUGetDataReadOnly(y_real);
        float* d_y_imag = (float*)mxGPUGetDataReadOnly(y_imag);

        for(int i = 0; i<2; i++)
        {
            mxGPUArray const *x_tmp;
            if(i==0) x_tmp = mxGPUCopyReal(x);
            if(i==1) x_tmp = mxGPUCopyImag(x);
            const float* const d_x = (float*)mxGPUGetDataReadOnly(x_tmp);
#if CUDART_VERSION >= 11000	
            if(i==0) cusparseStatus = cusparseXcsrmv_wrapper(cusparseHandle, trans, nrows, ncols, nnz, &alpha, descr, d_val, d_row_csr, d_col, d_x, &beta, d_y_real);
            if(i==1) cusparseStatus = cusparseXcsrmv_wrapper(cusparseHandle, trans, nrows, ncols, nnz, &alpha, descr, d_val, d_row_csr, d_col, d_x, &beta, d_y_imag);
#else
            if(i==0) cusparseStatus = hipsparseScsrmv(cusparseHandle, trans, nrows, ncols, nnz, &alpha, descr, d_val, d_row_csr, d_col, d_x, &beta, d_y_real);
            if(i==1) cusparseStatus = hipsparseScsrmv(cusparseHandle, trans, nrows, ncols, nnz, &alpha, descr, d_val, d_row_csr, d_col, d_x, &beta, d_y_imag);
#endif
            mxGPUDestroyGPUArray(x_tmp);
            if(cusparseStatus != HIPSPARSE_STATUS_SUCCESS) mxShowCriticalErrorMessage("csrmv failed.");
        }
        y = mxGPUCreateComplexGPUArray(y_real,y_imag);
        if (y==NULL) mxShowCriticalErrorMessage("mxGPUCreateComplexGPUArray failed.");
        mxGPUDestroyGPUArray(y_real);
        mxGPUDestroyGPUArray(y_imag);
    }
    else
    {
        const hipFloatComplex alpha = make_hipFloatComplex(1.0, 0.0);
        const hipFloatComplex beta = make_hipFloatComplex(0.0, 0.0);
        y = mxGPUCreateGPUArray(ndim, dims, cid, ccy, MX_GPU_INITIALIZE_VALUES);
        if (y==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed.");
        hipFloatComplex* d_y = (hipFloatComplex*)mxGPUGetData(y);
    	const hipFloatComplex* const d_val = (hipFloatComplex*)mxGPUGetDataReadOnly(val);
    	const hipFloatComplex* const d_x = (hipFloatComplex*)mxGPUGetDataReadOnly(x);
#if CUDART_VERSION >= 11000	
        cusparseStatus = cusparseXcsrmv_wrapper(cusparseHandle, trans, nrows, ncols, nnz, &alpha, descr, d_val, d_row_csr, d_col, d_x, &beta, d_y);
#else
        cusparseStatus = hipsparseCcsrmv(cusparseHandle, trans, nrows, ncols, nnz, &alpha, descr, d_val, d_row_csr, d_col, d_x, &beta, d_y);
#endif
    }
    
	// Return result
    if (cusparseStatus == HIPSPARSE_STATUS_SUCCESS)
    {
    	Y = mxGPUCreateMxArrayOnGPU(y);
    }
    else
    {
        mxShowCriticalErrorMessage("unknown failure",cusparseStatus);
    }

    // Clean up
    hipsparseDestroyMatDescr(descr);
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);
    mxGPUDestroyGPUArray(row_csr);
    mxGPUDestroyGPUArray(col);
    mxGPUDestroyGPUArray(val);
    mxGPUDestroyGPUArray(x);
    mxGPUDestroyGPUArray(y);
    mxFree(xdims);

    return;
}

