// 
// Mex wrapper to CUSPARSE format converter (csr2csc) to do transpose.
//
// Inspired by cusparse samples (conugateGradient) and Matlab gcsparse.
//  http://docs.nvidia.com/cuda/cusparse/index.html#cusparse-lt-t-gt-csrmv
//  http://www.mathworks.com/matlabcentral/fileexchange/44423-gpu-sparse--accumarray--non-uniform-grid
//

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

/* Using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

// Utilities and system includes
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper function CUDA error checking and initialization

// MATLAB related
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "mxShowCriticalErrorMessage.c"

// Input Arguments
#define	ROW_CSR prhs[0] // CSR format
#define	COL     prhs[1]
#define	VAL     prhs[2]
#define	NROWS   prhs[3]
#define	NCOLS   prhs[4]

// Output Arguments
#define	ROW     plhs[0]
#define	COL_CSC plhs[1] // CSC format
#define	VAL_CSC plhs[2]

void mexFunction(int nlhs, mxArray * plhs[], int nrhs, const mxArray * prhs[])
{
    // Checks
    if (nlhs > 3) mxShowCriticalErrorMessage("wrong number of output arguments");
    if (nrhs != 5) mxShowCriticalErrorMessage("wrong number of input arguments");

    // Initialize the MathWorks GPU API
    mxInitGPU();

    // Create Matlab pointers on the GPU
    mxGPUArray const *row_csr = mxGPUCreateFromMxArray(ROW_CSR);
    mxGPUArray const *col = mxGPUCreateFromMxArray(COL);
    mxGPUArray const *val = mxGPUCreateFromMxArray(VAL);

    // Checks - note rows must be in CSR format
    int nnz = mxGPUGetNumberOfElements(val);
    int nrows = (int)mxGetScalar(NROWS);
    int ncols = (int)mxGetScalar(NCOLS);
    if (mxGPUGetClassID(row_csr) != mxINT32_CLASS) mxShowCriticalErrorMessage("ROW_CSR argument is not int32");
    if (mxGPUGetClassID(col) != mxINT32_CLASS) mxShowCriticalErrorMessage("COL argument is not int32");
    if (mxGPUGetClassID(val) != mxSINGLE_CLASS) mxShowCriticalErrorMessage("VAL argument is not single");

    // Create space for output vectors
    const mwSize ndim = 1;
    mwSize dims[ndim];

    dims[0] = ncols+1;
    mxGPUArray *col_csc = mxGPUCreateGPUArray(ndim, dims, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    if (col_csc==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");

    dims[0] = nnz;
    mxGPUArray *row = mxGPUCreateGPUArray(ndim, dims, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    if (row==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");

    mxComplexity ccx = mxGPUGetComplexity(val);
    mxGPUArray *val_csc = mxGPUCreateGPUArray(ndim, dims, mxSINGLE_CLASS, ccx, MX_GPU_DO_NOT_INITIALIZE);
    if (val_csc==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");

    // Get handle to the CUBLAS context
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    checkCudaErrors(hipblasStatus_t);

    // Get handle to the CUSPARSE context
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&cusparseHandle);
    checkCudaErrors(cusparseStatus);
    hipsparseMatDescr_t descr = 0;
    cusparseStatus = hipsparseCreateMatDescr(&descr);
    checkCudaErrors(cusparseStatus);
    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ONE);

    // Convert from matlab pointers to native pointers 
    int *d_row_csr = (int*)mxGPUGetDataReadOnly(row_csr);
    int *d_col = (int*)mxGPUGetDataReadOnly(col);

    int *d_row = (int*)mxGPUGetData(row);
    int *d_col_csc = (int*)mxGPUGetData(col_csc);

    // Now we can access row_csr[] array
    int base;
    hipMemcpy(&base, d_row_csr, sizeof(int), hipMemcpyDeviceToHost);
    if (base != HIPSPARSE_INDEX_BASE_ONE) mxShowCriticalErrorMessage("ROW_CSR not using 1-based indexing");

    int nnz_check;
    hipMemcpy(&nnz_check, d_row_csr+nrows, sizeof(int), hipMemcpyDeviceToHost);
    nnz_check -= HIPSPARSE_INDEX_BASE_ONE;
    if (nnz_check != nnz) mxShowCriticalErrorMessage("ROW_CSR argument last element != nnz");

    // Convert from CSR to CSC
    hipsparseStatus_t status;

    if (ccx == mxREAL)
    {
    	float *d_val = (float*)mxGPUGetDataReadOnly(val);
    	float *d_val_csc = (float*)mxGPUGetData(val_csc);
	status =
  	hipsparseScsr2csc(cusparseHandle, nrows, ncols, nnz, d_val, d_row_csr, d_col, d_val_csc, d_row, d_col_csc, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ONE);
    }
    else
    {
    	hipComplex *d_val = (hipComplex*)mxGPUGetDataReadOnly(val);
    	hipComplex *d_val_csc = (hipComplex*)mxGPUGetData(val_csc);
	status =
  	hipsparseCcsr2csc(cusparseHandle, nrows, ncols, nnz, d_val, d_row_csr, d_col, d_val_csc, d_row, d_col_csc, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ONE);
    }

    if (status == HIPSPARSE_STATUS_SUCCESS)
    {
	// Return result
    	ROW = mxGPUCreateMxArrayOnGPU(row);
    	COL_CSC = mxGPUCreateMxArrayOnGPU(col_csc);
    	VAL_CSC = mxGPUCreateMxArrayOnGPU(val_csc);

    	// Make sure operations are finished before deleting
    	//hipDeviceSynchronize();
    }

    // Clean up
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);
    mxGPUDestroyGPUArray(val);
    mxGPUDestroyGPUArray(col);
    mxGPUDestroyGPUArray(row_csr);
    mxGPUDestroyGPUArray(val_csc);
    mxGPUDestroyGPUArray(col_csc);
    mxGPUDestroyGPUArray(row);

    // Failure
    if (status != HIPSPARSE_STATUS_SUCCESS)
    {
	char message[128];
	sprintf(message,"\nOperation hipsparseScsr2csc failed with error code %i",status);
	mxShowCriticalErrorMessage(message);
    }

    return;
}


