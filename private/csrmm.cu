// 
// Mex wrapper to CUSPARSE matrix-matrix multiply (csrmm).
//
// Inspired by cusparse samples (conugateGradient) and Matlab gcsparse.
//  http://docs.nvidia.com/cuda/cusparse/index.html#cusparse-lt-t-gt-csrmv
//  http://www.mathworks.com/matlabcentral/fileexchange/44423-gpu-sparse--accumarray--non-uniform-grid
//

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

/* Using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

// Utilities and system includes
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper function CUDA error checking and initialization

// MATLAB related
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "mxShowCriticalErrorMessage.c"

// Input Arguments
#define	ROW_CSR prhs[0] // this in CSR format (returned from coo2csr.cu)
#define	COL     prhs[1]
#define	VAL     prhs[2]
#define	NROWS   prhs[3]
#define	NCOLS   prhs[4]
#define	TRANS   prhs[5]
#define	B       prhs[6] // dense matrix

// Output Arguments
#define	C	plhs[0] // C = op(A) * B (sparse A, dense B)

void mexFunction(int nlhs, mxArray * plhs[], int nrhs, const mxArray * prhs[])
{
    // Checks
    if (nlhs > 1) mxShowCriticalErrorMessage("wrong number of output arguments");
    if (nrhs != 7) mxShowCriticalErrorMessage("wrong number of input arguments");

    if(mxIsGPUArray(ROW_CSR) == 0) mxShowCriticalErrorMessage("ROW_CSR argument is not on GPU");
    if(mxIsGPUArray(COL) == 0) mxShowCriticalErrorMessage("COL argument is not on GPU");
    if(mxIsGPUArray(VAL) == 0) mxShowCriticalErrorMessage("VAL argument is not on GPU");
    //if(mxIsGPUArray(B) == 0) mxShowCriticalErrorMessage("B argument is not on GPU");

    // Initialize the MathWorks GPU API
    mxInitGPU();

    // Create Matlab pointers on the GPU
    mxGPUArray const *row_csr = mxGPUCreateFromMxArray(ROW_CSR);
    mxGPUArray const *col = mxGPUCreateFromMxArray(COL);
    mxGPUArray const *val = mxGPUCreateFromMxArray(VAL);
    mxGPUArray const *b = mxGPUCreateFromMxArray(B);

    // Check sizes of A - note rows are in CSR (compressed row) format
    int nnz = mxGPUGetNumberOfElements(val);
    int m = (int)mxGetScalar(NROWS);
    int k = (int)mxGetScalar(NCOLS);

    if (mxGPUGetNumberOfElements(row_csr) != m+1) mxShowCriticalErrorMessage("ROW_CSR argument wrong size");
    if (mxGPUGetNumberOfElements(col) != nnz) mxShowCriticalErrorMessage("COL argument wrong size");

    // Check sizes of B
    if (mxGPUGetNumberOfDimensions(b) > 2) mxShowCriticalErrorMessage("B has too many dimensions");

    mwSize *bdims = (mwSize*)mxGPUGetDimensions(b); // dims always has >= 2 elements
    int ldb = bdims[0]; // leading dimension of B
    int n = bdims[1];

    hipsparseOperation_t trans = (hipsparseOperation_t)mxGetScalar(TRANS);
    if (trans == HIPSPARSE_OPERATION_NON_TRANSPOSE)
    {
	if (ldb != k) mxShowCriticalErrorMessage("B argument wrong size for multiply");
    }
    else
    {
	if (ldb != m) mxShowCriticalErrorMessage("B argument wrong size for transpose multiply");
    }

    // Check types
    if (mxGPUGetClassID(row_csr) != mxINT32_CLASS) mxShowCriticalErrorMessage("ROW_CSR argument is not int32");
    if (mxGPUGetClassID(col) != mxINT32_CLASS) mxShowCriticalErrorMessage("COL argument is not int32");
    if (mxGPUGetClassID(val) != mxSINGLE_CLASS) mxShowCriticalErrorMessage("VAL argument is not single");
    if (mxGPUGetClassID(b) != mxSINGLE_CLASS) mxShowCriticalErrorMessage("B argument is not single");

    // Check real/complex
    mxComplexity ccb = mxGPUGetComplexity(b);
    if (mxGPUGetComplexity(val) != ccb) mxShowCriticalErrorMessage("VAL and B must have same complexity");

    // Create space for output vectors
    const mwSize ndim = 2;
    mwSize cdims[ndim] = {trans == HIPSPARSE_OPERATION_NON_TRANSPOSE ? m : k, n};
    mxClassID cid = mxGPUGetClassID(b); // same class as B matrix
    int ldc = cdims[0]; // leading dimension of C
    
    mxGPUArray *c = mxGPUCreateGPUArray(ndim, cdims, cid, ccb, MX_GPU_INITIALIZE_VALUES); // initialize 0
    if (c==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed.");

    // Get handle to the CUBLAS context
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    checkCudaErrors(hipblasStatus_t);

    // Get handle to the CUSPARSE context
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&cusparseHandle);
    checkCudaErrors(cusparseStatus);
    hipsparseMatDescr_t descr = 0;
    cusparseStatus = hipsparseCreateMatDescr(&descr);
    checkCudaErrors(cusparseStatus);
    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ONE); // MATLAB unit offset

    // Convert from matlab pointers to native pointers 
    int *d_row_csr = (int*)mxGPUGetDataReadOnly(row_csr);
    int *d_col = (int*)mxGPUGetDataReadOnly(col);
    float *d_val = (float*)mxGPUGetDataReadOnly(val);
    float *d_b = (float*)mxGPUGetDataReadOnly(b);
    float *d_c = (float*)mxGPUGetData(c);

    // Now we can access the arrays, we can do some checks
    int base;
    hipMemcpy(&base, d_row_csr, sizeof(int), hipMemcpyDeviceToHost);
    if (base != HIPSPARSE_INDEX_BASE_ONE) mxShowCriticalErrorMessage("ROW_CSR not using 1-based indexing");

    int nnz_check;
    hipMemcpy(&nnz_check, d_row_csr+m, sizeof(int), hipMemcpyDeviceToHost);
    nnz_check -= HIPSPARSE_INDEX_BASE_ONE; // MATLAB unit offset
    if (nnz_check != nnz) mxShowCriticalErrorMessage("ROW_CSR argument last element != nnz");

    // Call cusparse multiply function in (S)ingle precision
    float alpha = 1.0;
    float beta = 0.0;

    hipsparseStatus_t status =
    hipsparseScsrmm(cusparseHandle, trans, m, n, k, nnz, &alpha, descr, d_val, d_row_csr, d_col, d_b, ldb, &beta, d_c, ldc);

    if (status == HIPSPARSE_STATUS_SUCCESS)
    {
    	// Return result
    	C = mxGPUCreateMxArrayOnGPU(c);

    	// Make sure operations are finished before deleting
    	//hipDeviceSynchronize();
    }

    // Clean up
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);
    mxGPUDestroyGPUArray(row_csr);
    mxGPUDestroyGPUArray(col);
    mxGPUDestroyGPUArray(val);
    mxGPUDestroyGPUArray(b);
    mxGPUDestroyGPUArray(c);
    mxFree(bdims);

    // Failure
    if (status != HIPSPARSE_STATUS_SUCCESS)
    {
	char message[128];
	sprintf(message,"\nOperation hipsparseScsrmm failed with error code %i",status);
	mxShowCriticalErrorMessage(message);
    }

    return;
}

