// 
// Mex wrapper to CUSPARSE format converter (csr2coo).
//
// Inspired by cusparse samples (conugateGradient) and Matlab gcsparse.
//  http://docs.nvidia.com/cuda/cusparse/index.html#cusparse-lt-t-gt-csrmv
//  http://www.mathworks.com/matlabcentral/fileexchange/44423-gpu-sparse--accumarray--non-uniform-grid
//

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

/* Using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

// Utilities and system includes
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper function CUDA error checking and initialization

// MATLAB related
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "mxShowCriticalErrorMessage.c"

// Input Arguments
#define	ROW_CSR prhs[0]
#define	NROWS   prhs[1]

// Output Arguments
#define	ROW plhs[0]

void mexFunction(int nlhs, mxArray * plhs[], int nrhs, const mxArray * prhs[])
{
    // Checks
    if (nlhs > 1) mxShowCriticalErrorMessage("wrong number of output arguments");
    if (nrhs != 2) mxShowCriticalErrorMessage("wrong number of input arguments");

    // Initialize the MathWorks GPU API
    mxInitGPU();

    // Create Matlab pointers on the GPU
    mxGPUArray const *row_csr = mxGPUCreateFromMxArray(ROW_CSR);

    // Checks - note rows must be in CSR format
    int nrows = (int)mxGetScalar(NROWS);
    if (mxGPUGetNumberOfElements(row_csr) != nrows+1) mxShowCriticalErrorMessage("ROW_CSR argument is wrong size");
    if (mxGPUGetClassID(row_csr) != mxINT32_CLASS) mxShowCriticalErrorMessage("ROW_CSR argument is not int32");

    // Get handle to the CUBLAS context
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    checkCudaErrors(hipblasStatus_t);

    // Get handle to the CUSPARSE context
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&cusparseHandle);
    checkCudaErrors(cusparseStatus);
    hipsparseMatDescr_t descr = 0;
    cusparseStatus = hipsparseCreateMatDescr(&descr);
    checkCudaErrors(cusparseStatus);
    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ONE);

    // Convert from matlab pointers to native pointers 
    int *d_row_csr = (int*)mxGPUGetDataReadOnly(row_csr);

    // Now we can access the arrays, we can do some checks
    int base;
    hipMemcpy(&base, d_row_csr, sizeof(int), hipMemcpyDeviceToHost);
    if (base != HIPSPARSE_INDEX_BASE_ONE) mxShowCriticalErrorMessage("ROW_CSR not using 1-based indexing");

    int nnz;
    hipMemcpy(&nnz, d_row_csr+nrows, sizeof(int), hipMemcpyDeviceToHost);
    nnz -= HIPSPARSE_INDEX_BASE_ONE;
    if (nnz < 0) mxShowCriticalErrorMessage("ROW_CSR returned negative nnz");

    // Create space for output vector
    const mwSize ndim = 1;
    mwSize dims[ndim] = {nnz};
    mxGPUArray *row = mxGPUCreateGPUArray(ndim, dims, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    if (row==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");

    // Convert from matlab pointers to native pointers 
    int *d_row = (int*)mxGPUGetData(row);

    // Call csr2coo
    hipsparseStatus_t status =
    hipsparseXcsr2coo(cusparseHandle, d_row_csr, nnz, nrows, d_row, HIPSPARSE_INDEX_BASE_ONE);

    if (status == HIPSPARSE_STATUS_SUCCESS)
    {
    	// Return result
    	ROW = mxGPUCreateMxArrayOnGPU(row);

    	// Make sure operations are finished before deleting
    	//hipDeviceSynchronize();
    }

    // Clean up
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);
    mxGPUDestroyGPUArray(row);
    mxGPUDestroyGPUArray(row_csr);

    // Failure
    if (status != HIPSPARSE_STATUS_SUCCESS)
    {
	char message[128];
	sprintf(message,"\nOperation hipsparseXcsr2coo failed with error code %i.",status);
	mxShowCriticalErrorMessage(message);
    }

    return;
}
