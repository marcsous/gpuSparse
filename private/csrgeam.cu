// 
// Mex wrapper to CUSPARSE matrix-matrix addition (csrgeam).
//
// Inspired by cusparse samples (conugateGradient) and Matlab gcsparse.
//  http://docs.nvidia.com/cuda/cusparse/index.html#cusparse-lt-t-gt-csrmv
//  http://www.mathworks.com/matlabcentral/fileexchange/44423-gpu-sparse--accumarray--non-uniform-grid
//

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

/* Using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

#if CUDART_VERSION >= 11000
#include "wrappers_to_cuda_11.h"
#endif    
        
// MATLAB related
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "mxShowCriticalErrorMessage.c"

// Input Arguments
#define	A_ROW_CSR prhs[0] // this in CSR format (returned from coo2csr.cu)
#define	A_COL     prhs[1]
#define	A_VAL     prhs[2]
#define	NROWS     prhs[3]
#define	NCOLS     prhs[4]
#define	B_ROW_CSR prhs[5] // this in CSR format (returned from coo2csr.cu)
#define	B_COL     prhs[6]
#define	B_VAL     prhs[7]
#define	ALPHA     prhs[8] // scalar: C = ALPHA*A + BETA*B
#define	BETA      prhs[9] // scalar: C = ALPHA*A + BETA*B

// Output Arguments
#define	C_ROW_CSR plhs[0] // this in CSR format (returned from coo2csr.cu)
#define	C_COL     plhs[1]
#define	C_VAL     plhs[2]

void mexFunction(int nlhs, mxArray * plhs[], int nrhs, const mxArray * prhs[])
{
    // Checks
    if (nlhs > 3) mxShowCriticalErrorMessage("wrong number of output arguments",nlhs);
    if (nrhs != 10) mxShowCriticalErrorMessage("wrong number of input arguments",nrhs);

    if(!mxIsGPUArray(A_ROW_CSR)) mxShowCriticalErrorMessage("A_ROW_CSR argument is not on GPU");
    if(!mxIsGPUArray(A_COL)) mxShowCriticalErrorMessage("A_COL argument is not on GPU");
    if(!mxIsGPUArray(A_VAL)) mxShowCriticalErrorMessage("A_VAL argument is not on GPU");

    if (!mxIsScalar(ALPHA)) mxShowCriticalErrorMessage("ALPHA argument must be a scalar");
    if (!mxIsScalar(BETA)) mxShowCriticalErrorMessage("BETA argument must be a scalar");
    if (!mxIsScalar(NROWS)) mxShowCriticalErrorMessage("NROWS argument must be a scalar");
    if (!mxIsScalar(NCOLS)) mxShowCriticalErrorMessage("NCOLS argument must be a scalar");

    // Initialize the MathWorks GPU API
    mxInitGPU();

    // Create Matlab pointers on the GPU
    mxGPUArray const *a_row_csr = mxGPUCreateFromMxArray(A_ROW_CSR);
    mxGPUArray const *a_col = mxGPUCreateFromMxArray(A_COL);
    mxGPUArray const *a_val = mxGPUCreateFromMxArray(A_VAL);
    mxGPUArray const *b_row_csr = mxGPUCreateFromMxArray(B_ROW_CSR);
    mxGPUArray const *b_col = mxGPUCreateFromMxArray(B_COL);
    mxGPUArray const *b_val = mxGPUCreateFromMxArray(B_VAL);

    // Check sizes - note rows are in CSR (compressed row) format
    int a_nnz = mxGPUGetNumberOfElements(a_val);
    int b_nnz = mxGPUGetNumberOfElements(b_val);

    mwSize nrows = mxGetScalar(NROWS);
    mwSize ncols = mxGetScalar(NCOLS);

    if (mxGPUGetNumberOfElements(a_row_csr) != nrows+1) mxShowCriticalErrorMessage("A_ROW_CSR argument wrong size",mxGPUGetNumberOfElements(a_row_csr));
    if (mxGPUGetNumberOfElements(a_col) != a_nnz) mxShowCriticalErrorMessage("A_COL argument wrong size",mxGPUGetNumberOfElements(a_col));

    if (mxGPUGetNumberOfElements(b_row_csr) != nrows+1) mxShowCriticalErrorMessage("B_ROW_CSR argument wrong size",mxGPUGetNumberOfElements(b_row_csr));
    if (mxGPUGetNumberOfElements(b_col) != b_nnz) mxShowCriticalErrorMessage("B_COL argument wrong size",mxGPUGetNumberOfElements(b_col));

    if (mxGPUGetClassID(a_row_csr) != mxINT32_CLASS) mxShowCriticalErrorMessage("A_ROW_CSR argument is not int32");
    if (mxGPUGetClassID(a_col) != mxINT32_CLASS) mxShowCriticalErrorMessage("A_COL argument is not int32");
    if (mxGPUGetClassID(a_val) != mxSINGLE_CLASS) mxShowCriticalErrorMessage("A_VAL argument is not single");

    if (mxGPUGetClassID(b_row_csr) != mxINT32_CLASS) mxShowCriticalErrorMessage("B_ROW argument is not int32");
    if (mxGPUGetClassID(b_col) != mxINT32_CLASS) mxShowCriticalErrorMessage("B_COL argument is not int32");
    if (mxGPUGetClassID(b_val) != mxSINGLE_CLASS) mxShowCriticalErrorMessage("B_VAL argument is not single");

    // Allocate space for output row vector
    const mwSize ndim = 1;
    mwSize dims[ndim] = {nrows+1};
    mxGPUArray *c_row_csr = mxGPUCreateGPUArray(ndim, dims, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    if (c_row_csr==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed.");

    // Get handle to the CUBLAS context
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) mxShowCriticalErrorMessage(hipblasStatus_t);

    // Get handle to the CUSPARSE context
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&cusparseHandle);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) mxShowCriticalErrorMessage(cusparseStatus);
    hipsparseMatDescr_t descr = 0;
    cusparseStatus = hipsparseCreateMatDescr(&descr);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) mxShowCriticalErrorMessage(cusparseStatus);
    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ONE);

    // Convert from matlab pointers to native pointers
    const int* const d_a_col = (int*)mxGPUGetDataReadOnly(a_col);
    const int* const d_b_col = (int*)mxGPUGetDataReadOnly(b_col);

    const float* const d_a_val = (float*)mxGPUGetDataReadOnly(a_val);
    const float* const d_b_val = (float*)mxGPUGetDataReadOnly(b_val);

    const int* const d_a_row_csr = (int*)mxGPUGetDataReadOnly(a_row_csr);
    const int* const d_b_row_csr = (int*)mxGPUGetDataReadOnly(b_row_csr);

    int *d_c_col = NULL;
    float *d_c_val = NULL;
    int *d_c_row_csr = (int*)mxGPUGetData(c_row_csr);
   
    const float alpha = (float)mxGetScalar(ALPHA);
    const float beta = (float)mxGetScalar(BETA);

    // Now we can access the arrays, we can do some checks
    int base;
    hipMemcpy(&base, d_a_row_csr, sizeof(int), hipMemcpyDeviceToHost);
    if (base != HIPSPARSE_INDEX_BASE_ONE) mxShowCriticalErrorMessage("A_ROW_CSR not using 1-based indexing");

    int nnz_check;
    hipMemcpy(&nnz_check, d_a_row_csr+nrows, sizeof(int), hipMemcpyDeviceToHost);
    nnz_check -= HIPSPARSE_INDEX_BASE_ONE;
    if (nnz_check != a_nnz) mxShowCriticalErrorMessage("A_ROW_CSR argument last element != nnz",nnz_check);

    hipMemcpy(&base, d_b_row_csr, sizeof(int), hipMemcpyDeviceToHost);
    if (base != HIPSPARSE_INDEX_BASE_ONE) mxShowCriticalErrorMessage("B_ROW_CSR not using 1-based indexing");

    hipMemcpy(&nnz_check, d_b_row_csr+nrows, sizeof(int), hipMemcpyDeviceToHost);
    nnz_check -= HIPSPARSE_INDEX_BASE_ONE;
    if (nnz_check != b_nnz) mxShowCriticalErrorMessage("B_ROW_CSR argument last element != nnz",nnz_check);

    // Get sparsity pattern and nnz of output matrix
    int c_nnz;
    int *nnzTotalDevHostPtr = &c_nnz;
    hipsparseSetPointerMode(cusparseHandle, HIPSPARSE_POINTER_MODE_HOST);

    char *buffer = NULL;            
    size_t bufferSizeInBytes;

#if CUDART_VERSION >= 11000
    hipsparseScsrgeam2_bufferSizeExt(cusparseHandle, nrows, ncols,
        &alpha,
        descr, a_nnz, d_a_val, d_a_row_csr, d_a_col,
        &beta,
        descr, b_nnz, d_b_val, d_b_row_csr, d_b_col,
        descr,        d_c_val, d_c_row_csr, d_c_col,
        &bufferSizeInBytes);

    hipError_t status0 = hipMalloc((void**)&buffer, sizeof(char)*bufferSizeInBytes);
    if (status0 != hipSuccess)
    {
        mxShowCriticalErrorMessage("Operation hipMalloc failed",status0);
    }

    hipsparseStatus_t status1 =
    hipsparseXcsrgeam2Nnz(cusparseHandle, nrows, ncols,
        	descr, a_nnz, d_a_row_csr, d_a_col,
        	descr, b_nnz, d_b_row_csr, d_b_col,
        	descr, d_c_row_csr, nnzTotalDevHostPtr, buffer);
#else
    hipsparseStatus_t status1 =
    hipsparseXcsrgeamNnz(cusparseHandle, nrows, ncols,
        	descr, a_nnz, d_a_row_csr, d_a_col,
        	descr, b_nnz, d_b_row_csr, d_b_col,
        	descr, d_c_row_csr, nnzTotalDevHostPtr);
#endif

    // Failure
    if (status1 != HIPSPARSE_STATUS_SUCCESS)
    {
        mxShowCriticalErrorMessage("Operation hipsparseXcsrgeamNnz failed",status1);
    }

    if (NULL != nnzTotalDevHostPtr)
    {
        c_nnz = *nnzTotalDevHostPtr;
    }
    else
    {
    	int baseC = HIPSPARSE_INDEX_BASE_ONE;
        hipMemcpy(&c_nnz, d_c_row_csr+nrows, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&baseC, c_row_csr, sizeof(int), hipMemcpyDeviceToHost);
        c_nnz -= baseC;
    }

    // Allocate space for output vectors
    dims[0] = {(mwSize)c_nnz};
    mxGPUArray *c_col = mxGPUCreateGPUArray(ndim, dims, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    if (c_col==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");

    mxGPUArray *c_val = mxGPUCreateGPUArray(ndim, dims, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
    if (c_val==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");

    // Convert from matlab pointers to native pointers
    d_c_col = (int*)mxGPUGetData(c_col);
    d_c_val = (float*)mxGPUGetData(c_val);

    // Addition here
#if CUDART_VERSION >= 11000
    hipsparseStatus_t status2 =
    hipsparseScsrgeam2(cusparseHandle, nrows, ncols,
	        &alpha,
	        descr, a_nnz,
	        d_a_val, d_a_row_csr, d_a_col,
	        &beta,
	        descr, b_nnz,
	        d_b_val, d_b_row_csr, d_b_col,
	        descr,
	        d_c_val, d_c_row_csr, d_c_col, buffer);
#else
    hipsparseStatus_t status2 =
    hipsparseScsrgeam(cusparseHandle, nrows, ncols,
	        &alpha,
	        descr, a_nnz,
	        d_a_val, d_a_row_csr, d_a_col,
	        &beta,
	        descr, b_nnz,
	        d_b_val, d_b_row_csr, d_b_col,
	        descr,
	        d_c_val, d_c_row_csr, d_c_col);
#endif

    if (status2 == HIPSPARSE_STATUS_SUCCESS)
    {
    	// Return results
    	C_ROW_CSR = mxGPUCreateMxArrayOnGPU(c_row_csr);
    	C_COL = mxGPUCreateMxArrayOnGPU(c_col);
    	C_VAL = mxGPUCreateMxArrayOnGPU(c_val);

    	// Make sure operations are finished before deleting
    	//hipDeviceSynchronize();
    }

    // Clean up
    hipsparseDestroyMatDescr(descr);
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);
    if(buffer) hipFree(buffer);
    mxGPUDestroyGPUArray(a_row_csr);
    mxGPUDestroyGPUArray(a_col);
    mxGPUDestroyGPUArray(a_val);
    mxGPUDestroyGPUArray(b_row_csr);
    mxGPUDestroyGPUArray(b_col);
    mxGPUDestroyGPUArray(b_val);
    mxGPUDestroyGPUArray(c_row_csr);
    mxGPUDestroyGPUArray(c_col);
    mxGPUDestroyGPUArray(c_val);

    // Failure
    if (status2 != HIPSPARSE_STATUS_SUCCESS)
    {
	mxShowCriticalErrorMessage("Operation hipsparseScsrgeam failed",status2);
    }

    return;
}
