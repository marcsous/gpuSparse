// 
// Mex wrapper to CUSPARSE sort for CSR format (csrsort).
//
// Inspired by cusparse samples (conugateGradient) and Matlab gcsparse.
//  http://docs.nvidia.com/cuda/cusparse/index.html#cusparse-lt-t-gt-csrmv
//  http://www.mathworks.com/matlabcentral/fileexchange/44423-gpu-sparse--accumarray--non-uniform-grid
//

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

/* Using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

// MATLAB related
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "mxShowCriticalErrorMessage.c"

// Input Arguments
#define	ROW_CSR prhs[0]
#define	COL     prhs[1]
#define	VAL     prhs[2]
#define	NROWS   prhs[3]
#define	NCOLS   prhs[4]

// Output Arguments
#define	COL_SORT plhs[0]
#define	VAL_SORT plhs[1]

void mexFunction(int nlhs, mxArray * plhs[], int nrhs, const mxArray * prhs[])
{
    // Checks
    if (nlhs > 2) mxShowCriticalErrorMessage("wrong number of output arguments",nlhs);
    if (nrhs != 5) mxShowCriticalErrorMessage("wrong number of input arguments",nrhs);

    // Initialize the MathWorks GPU API
    mxInitGPU();

    // Create Matlab pointers on the GPU
    mxGPUArray const *row_csr = mxGPUCreateFromMxArray(ROW_CSR);
    mxGPUArray const *col = mxGPUCreateFromMxArray(COL);
    mxGPUArray const *val = mxGPUCreateFromMxArray(VAL);

    // Checks - note vectors must be in CSR format
    int nnz = mxGPUGetNumberOfElements(val);
    if (mxGPUGetNumberOfElements(col) != nnz) mxShowCriticalErrorMessage("COL and VAL argument length mismatch");

    if (!mxIsScalar(NROWS)) mxShowCriticalErrorMessage("NROWS argument must be a scalar");
    if (!mxIsScalar(NCOLS)) mxShowCriticalErrorMessage("NCOLS argument must be a scalar");

    int ncols = (int)mxGetScalar(NCOLS);
    int nrows = (int)mxGetScalar(NROWS);
    if (mxGPUGetNumberOfElements(row_csr) != nrows+1) mxShowCriticalErrorMessage("ROW_CSR argument wrong size");

    if (mxGPUGetClassID(row_csr) != mxINT32_CLASS) mxShowCriticalErrorMessage("ROW_CSR argument is not int32");
    if (mxGPUGetClassID(col) != mxINT32_CLASS) mxShowCriticalErrorMessage("COL argument is not int32");
    if (mxGPUGetClassID(val) != mxSINGLE_CLASS) mxShowCriticalErrorMessage("VAL argument is not single");

    // Create space for output vectors
    const mwSize ndim = 1;
    mwSize dims[ndim];

    dims[0] = nnz;
    mxGPUArray *col_sort = mxGPUCreateGPUArray(ndim, dims, mxINT32_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    if (col_sort==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");

    mxComplexity ccx = mxGPUGetComplexity(val);
    mxGPUArray *val_sort = mxGPUCreateGPUArray(ndim, dims, mxSINGLE_CLASS, ccx, MX_GPU_INITIALIZE_VALUES);
    if (val_sort==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");

    // Get handle to the CUBLAS context
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) mxShowCriticalErrorMessage(hipblasStatus_t);

    // Get handle to the CUSPARSE context
    hipError_t cudaStatus;
    hipsparseStatus_t cusparseStatus;
    hipsparseHandle_t cusparseHandle = 0;
    cusparseStatus = hipsparseCreate(&cusparseHandle);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) mxShowCriticalErrorMessage(cusparseStatus);
    hipsparseMatDescr_t descr = 0;
    cusparseStatus = hipsparseCreateMatDescr(&descr);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) mxShowCriticalErrorMessage(cusparseStatus);
    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ONE);

    // Convert from matlab pointers to native pointers 
    const int * const d_row_csr = (int*)mxGPUGetDataReadOnly(row_csr);
    const int * const d_col = (int*)mxGPUGetDataReadOnly(col);
    int *d_col_sort = (int*)mxGPUGetData(col_sort);

   // Now we can access the arrays, we can do some checks
    int base;
    hipMemcpy(&base, d_row_csr, sizeof(int), hipMemcpyDeviceToHost);
    if (base != HIPSPARSE_INDEX_BASE_ONE) mxShowCriticalErrorMessage("A_ROW_CSR not using 1-based indexing");

    int nnz_check;
    hipMemcpy(&nnz_check, d_row_csr+nrows, sizeof(int), hipMemcpyDeviceToHost);
    nnz_check -= HIPSPARSE_INDEX_BASE_ONE;
    if (nnz_check != nnz) mxShowCriticalErrorMessage("ROW_CSR argument last element != nnz",nnz_check);

    // Since sort is in-place, copy the read-only vectors to read-write ones
    cudaStatus = hipMemcpy((void *)d_col_sort, d_col, nnz*sizeof(int), hipMemcpyDeviceToDevice);
    if (cudaStatus != hipSuccess) mxShowCriticalErrorMessage("Operation hipMemcpy failed");

    if (ccx == mxREAL)
    {
    	const float * const d_val = (float*)mxGPUGetDataReadOnly(val);
    	float *d_val_sort = (float*)mxGPUGetData(val_sort);
    	cudaStatus = hipMemcpy((void *)d_val_sort, d_val, nnz*sizeof(float), hipMemcpyDeviceToDevice);
    }
    else
    {
    	const hipFloatComplex * const d_val = (hipFloatComplex*)mxGPUGetDataReadOnly(val);
    	hipFloatComplex *d_val_sort = (hipFloatComplex*)mxGPUGetData(val_sort);
    	cudaStatus = hipMemcpy((void *)d_val_sort, d_val, nnz*sizeof(hipFloatComplex), hipMemcpyDeviceToDevice);
    }
    if (cudaStatus != hipSuccess) mxShowCriticalErrorMessage("Operation hipMemcpy failed",cudaStatus);

    // Sort by rows
    int *P = NULL;
    void *pBuffer = NULL;
    size_t pBufferSizeInBytes = 0;

    if (nnz > 0)
    {
    	// step 1: allocate buffer
    	cusparseStatus = hipsparseXcsrsort_bufferSizeExt(cusparseHandle, nrows, ncols, nnz, d_row_csr, d_col, &pBufferSizeInBytes);
    	if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) mxShowCriticalErrorMessage("Operation hipsparseXcoosort_bufferSizeExt failed",cusparseStatus);

    	cudaStatus = hipMalloc( &pBuffer, sizeof(char)*pBufferSizeInBytes);
    	if (cudaStatus != hipSuccess) mxShowCriticalErrorMessage("Operation hipMalloc failed",cudaStatus);

    	// step 2: setup permutation vector P to identity
    	cudaStatus = hipMalloc( &P, sizeof(int)*nnz);
    	if (cudaStatus != hipSuccess) mxShowCriticalErrorMessage("Operation hipMalloc failed",cudaStatus);

	cusparseStatus = hipsparseCreateIdentityPermutation(cusparseHandle, nnz, P);
    	if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) mxShowCriticalErrorMessage("Operation hipsparseCreateIdentityPermutation failed",cusparseStatus);

    	// step 3: sort COO format by Row
    	cusparseStatus = hipsparseXcsrsort(cusparseHandle, nrows, ncols, nnz, descr, d_row_csr, d_col_sort, P, pBuffer);
    	if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) mxShowCriticalErrorMessage("Operation hipsparseXcsrsort failed",cusparseStatus);

    	// step 4: gather sorted cooVals
    	if (ccx == mxREAL)
    	{
    	    const float * const d_val = (float*)mxGPUGetDataReadOnly(val);
    	    float *d_val_sort = (float*)mxGPUGetData(val_sort);
	    cusparseStatus = hipsparseSgthr(cusparseHandle, nnz, d_val, d_val_sort, P, HIPSPARSE_INDEX_BASE_ZERO); // MUST USE BASE_ZERO
	}
	else
	{
    	    const hipFloatComplex * const d_val = (hipFloatComplex*)mxGPUGetDataReadOnly(val);
    	    hipFloatComplex *d_val_sort = (hipFloatComplex*)mxGPUGetData(val_sort);
	    cusparseStatus = hipsparseCgthr(cusparseHandle, nnz, d_val, d_val_sort, P, HIPSPARSE_INDEX_BASE_ZERO); // MUST USE BASE_ZERO
	}
    	if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) mxShowCriticalErrorMessage("Operation hipsparseSgthr or hipsparseCgthr failed",cusparseStatus);

    }

    // Return result
    COL_SORT = mxGPUCreateMxArrayOnGPU(col_sort);
    VAL_SORT = mxGPUCreateMxArrayOnGPU(val_sort);

    // Make sure operations are finished before deleting
    //hipDeviceSynchronize();

    // Clean up
    hipsparseDestroyMatDescr(descr);
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);
    mxGPUDestroyGPUArray(row_csr);
    mxGPUDestroyGPUArray(col);
    mxGPUDestroyGPUArray(col_sort);
    mxGPUDestroyGPUArray(val);
    mxGPUDestroyGPUArray(val_sort);
    if (pBuffer) hipFree(pBuffer);
    if (P) hipFree(P);

    return;
}
