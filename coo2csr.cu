// 
// Mex wrapper to CUSPARSE format converter (coo2csr).
//
// Inspired by cusparse samples (conugateGradient) and Matlab gcsparse.
//  http://docs.nvidia.com/cuda/cusparse/index.html#cusparse-lt-t-gt-csrmv
//  http://www.mathworks.com/matlabcentral/fileexchange/44423-gpu-sparse--accumarray--non-uniform-grid
//

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

/* Using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

// MATLAB related
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "mxShowCriticalErrorMessage.c"

// Input Arguments
#define	ROW   prhs[0]
#define	NROWS prhs[1]

// Output Arguments
#define	ROW_CSR plhs[0]

void mexFunction(int nlhs, mxArray * plhs[], int nrhs, const mxArray * prhs[])
{
    // Checks
    if (nlhs > 1) mxShowCriticalErrorMessage("wrong number of output arguments",nlhs);
    if (nrhs != 2) mxShowCriticalErrorMessage("wrong number of input arguments",nrhs);

    // Initialize the MathWorks GPU API
    mxInitGPU();

    // Create Matlab pointers on the GPU
    mxGPUArray const *row = mxGPUCreateFromMxArray(ROW);

    // Checks - note rows must be in COO (uncompressed) format
    if (!mxIsScalar(NROWS)) mxShowCriticalErrorMessage("NROWS argument must be a scalar");
    if (mxGPUGetClassID(row) != mxINT32_CLASS) mxShowCriticalErrorMessage("ROW argument is not int32");

    mwSize nrows = mxGetScalar(NROWS);
    mwSize nnz = mxGPUGetNumberOfElements(row);

    // Create space for output vector
    const mwSize ndim = 1;
    mwSize dims[ndim] = {nrows+1};
    mxGPUArray *row_csr = mxGPUCreateGPUArray(ndim, dims, mxINT32_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
    if (row_csr==NULL) mxShowCriticalErrorMessage("mxGPUCreateGPUArray failed");

    // Get handle to the CUBLAS context
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) mxShowCriticalErrorMessage(hipblasStatus_t);

    // Get handle to the CUSPARSE context
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&cusparseHandle);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) mxShowCriticalErrorMessage(cusparseStatus);
    hipsparseMatDescr_t descr = 0;
    cusparseStatus = hipsparseCreateMatDescr(&descr);
    if (cusparseStatus != HIPSPARSE_STATUS_SUCCESS) mxShowCriticalErrorMessage(cusparseStatus);
    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ONE);

    // Convert from matlab pointers to native pointers 
    const int * const d_row = (int*)mxGPUGetDataReadOnly(row);
    int *d_row_csr = (int*)mxGPUGetData(row_csr);
    char message[128] = {'\0'};
    int *buffer = NULL;

    // Call coo2csr - returns uninitialized when nnz==0 so need to handle separately
    if (nnz == 0)
    {
	buffer = (int *)mxMalloc((nrows+1)*sizeof(int));
	if (buffer == NULL) mxShowCriticalErrorMessage("mxMalloc failed");
    	for (int j=0; j<nrows+1; j++) buffer[j] = HIPSPARSE_INDEX_BASE_ONE;
	hipError_t status = hipMemcpy((void *)d_row_csr, buffer, (nrows+1)*sizeof(int), hipMemcpyHostToDevice);
	if (status != hipSuccess) sprintf(message,"\nOperation hipMemcpy failed with error code %i",status);
    }
    else
    {
    	hipsparseStatus_t status = hipsparseXcoo2csr(cusparseHandle, d_row, nnz, nrows, d_row_csr, HIPSPARSE_INDEX_BASE_ONE);
    	if (status != HIPSPARSE_STATUS_SUCCESS) sprintf(message,"\nOperation hipsparseXcoo2csr failed with error code %i",status);
    }


    if (message[0] == '\0')
    {
    	// Return result
    	ROW_CSR = mxGPUCreateMxArrayOnGPU(row_csr);

    	// Make sure operations are finished before deleting
    	//hipDeviceSynchronize();
    }

    // Clean up
    hipsparseDestroyMatDescr(descr);
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);
    mxGPUDestroyGPUArray(row);
    mxGPUDestroyGPUArray(row_csr);
    if (buffer) mxFree(buffer);

    // Failure
    if (message[0] != '\0') mxShowCriticalErrorMessage(message);

    return;
}
